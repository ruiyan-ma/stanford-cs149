#include "hip/hip_runtime.h"
#include <stdio.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <driver_functions.h>

#include <thrust/scan.h>
#include <thrust/device_ptr.h>
#include <thrust/device_malloc.h>
#include <thrust/device_free.h>

#include "CycleTimer.h"

#define THREADS_PER_BLOCK 256


// helper function to round an integer up to the next power of 2
static inline int nextPow2(int n) {
    n--;
    n |= n >> 1;
    n |= n >> 2;
    n |= n >> 4;
    n |= n >> 8;
    n |= n >> 16;
    n++;
    return n;
}

__global__ void
upsweep_knl(int* output, int N, int two_d) {
    int two_dplus1 = 2 * two_d;
    int i = (blockIdx.x * blockDim.x + threadIdx.x) * two_dplus1;

    if (i < N) {
        output[i + two_dplus1 - 1] += output[i + two_d - 1];
    }
}

__global__ void
downsweep_knl(int* output, int N, int two_d) {
    int two_dplus1 = 2 * two_d;
    int i = (blockIdx.x * blockDim.x + threadIdx.x) * two_dplus1;

    if (i < N) {
        int temp = output[i + two_d - 1];
        output[i + two_d - 1] = output[i + two_dplus1 - 1];
        output[i + two_dplus1 - 1] += temp;
    }
}

__global__ void
set_zero_knl(int* output, int index) {
    output[index] = 0;
}

// exclusive_scan --
//
// Implementation of an exclusive scan on global memory array `input`,
// with results placed in global memory `result`.
//
// N is the logical size of the input and output arrays, however
// students can assume that both the start and result arrays we
// allocated with next power-of-two sizes as described by the comments
// in cudaScan().  This is helpful, since your parallel scan
// will likely write to memory locations beyond N, but of course not
// greater than N rounded up to the next power of 2.
//
// Also, as per the comments in cudaScan(), you can implement an
// "in-place" scan, since the timing harness makes a copy of input and
// places it in result
void exclusive_scan(int* input, int N, int* result)
{

    // CS149 TODO:
    //
    // Implement your exclusive scan implementation here.  Keep in
    // mind that although the arguments to this function are device
    // allocated arrays, this is a function that is running in a thread
    // on the CPU.  Your implementation will need to make multiple calls
    // to CUDA kernel functions (that you must write) to implement the
    // scan.

    // set N to the next power-of-two
    N = nextPow2(N);

    // upsweep phase
    for (int two_d = 1; two_d <= N / 2; two_d *= 2) {
        int thread_num = N / (2 * two_d);
        int blocks = (thread_num + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
        int threadsPerBlock = blocks == 1 ? thread_num : THREADS_PER_BLOCK;
        upsweep_knl<<<blocks, threadsPerBlock>>>(result, N, two_d);
        hipDeviceSynchronize();
    }

    set_zero_knl<<<1, 1>>>(result, N - 1);
    hipDeviceSynchronize();

    // downsweep phase
    for (int two_d = N / 2; two_d >= 1; two_d /= 2) {
        int thread_num = N / (2 * two_d);
        int blocks = (thread_num + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
        int threadsPerBlock = blocks == 1 ? thread_num : THREADS_PER_BLOCK;
        downsweep_knl<<<blocks, threadsPerBlock>>>(result, N, two_d);
        hipDeviceSynchronize();
    }
}


//
// cudaScan --
//
// This function is a timing wrapper around the student's
// implementation of scan - it copies the input to the GPU
// and times the invocation of the exclusive_scan() function
// above. Students should not modify it.
double cudaScan(int* inarray, int* end, int* resultarray)
{
    int* device_result;
    int* device_input;
    int N = end - inarray;  

    // This code rounds the arrays provided to exclusive_scan up
    // to a power of 2, but elements after the end of the original
    // input are left uninitialized and not checked for correctness.
    //
    // Student implementations of exclusive_scan may assume an array's
    // allocated length is a power of 2 for simplicity. This will
    // result in extra work on non-power-of-2 inputs, but it's worth
    // the simplicity of a power of two only solution.

    int rounded_length = nextPow2(end - inarray);
    
    hipMalloc((void **)&device_result, sizeof(int) * rounded_length);
    hipMalloc((void **)&device_input, sizeof(int) * rounded_length);

    // For convenience, both the input and output vectors on the
    // device are initialized to the input values. This means that
    // students are free to implement an in-place scan on the result
    // vector if desired.  If you do this, you will need to keep this
    // in mind when calling exclusive_scan from find_repeats.
    hipMemcpy(device_input, inarray, (end - inarray) * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(device_result, inarray, (end - inarray) * sizeof(int), hipMemcpyHostToDevice);

    double startTime = CycleTimer::currentSeconds();

    exclusive_scan(device_input, N, device_result);

    // Wait for completion
    hipDeviceSynchronize();
    double endTime = CycleTimer::currentSeconds();
       
    hipMemcpy(resultarray, device_result, (end - inarray) * sizeof(int), hipMemcpyDeviceToHost);

    double overallDuration = endTime - startTime;
    return overallDuration; 
}


// cudaScanThrust --
//
// Wrapper around the Thrust library's exclusive scan function
// As above in cudaScan(), this function copies the input to the GPU
// and times only the execution of the scan itself.
//
// Students are not expected to produce implementations that achieve
// performance that is competition to the Thrust version, but it is fun to try.
double cudaScanThrust(int* inarray, int* end, int* resultarray) {

    int length = end - inarray;
    thrust::device_ptr<int> d_input = thrust::device_malloc<int>(length);
    thrust::device_ptr<int> d_output = thrust::device_malloc<int>(length);
    
    hipMemcpy(d_input.get(), inarray, length * sizeof(int), hipMemcpyHostToDevice);

    double startTime = CycleTimer::currentSeconds();

    thrust::exclusive_scan(d_input, d_input + length, d_output);

    hipDeviceSynchronize();
    double endTime = CycleTimer::currentSeconds();
   
    hipMemcpy(resultarray, d_output.get(), length * sizeof(int), hipMemcpyDeviceToHost);

    thrust::device_free(d_input);
    thrust::device_free(d_output);

    double overallDuration = endTime - startTime;
    return overallDuration; 
}

// set_flag_knl: set flag when input[i] == input[i + 1]
//
// input:  [1, 2, 2, 1, 1, 1, 3, 5, 3, 3]
// output: [0, 1, 0, 1, 1, 0, 0, 0, 1, 0]
__global__ void
set_flag_knl(int* input, int* output, int length) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < length - 1) {
        output[index] = input[index] == input[index + 1] ? 1 : 0;
    }
}

// collect_index_knl: when flags[i] == 1, output.push_back(i)
//
// index:  [0, 1, 2, 3, 4, 5, 6, 7, 8, 9]
// flags:  [0, 1, 0, 1, 1, 0, 0, 0, 1, 0]
// prefix: [_, 0, _, 1, 2, _, _, _, 3, _]
// output: [1, 3, 4, 8]
__global__ void
collect_index_knl(int* flags, int* prefix, int* output, int length) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < length - 1) {
        if (flags[index] == 1) {
            output[prefix[index]] = index;
        }
    }
}

// find_repeats --
//
// Given an array of integers `device_input`, returns an array of all
// indices `i` for which `device_input[i] == device_input[i+1]`.
//
// Returns the total number of pairs found
int find_repeats(int* device_input, int length, int* device_output) {

    // CS149 TODO:
    //
    // Implement this function. You will probably want to
    // make use of one or more calls to exclusive_scan(), as well as
    // additional CUDA kernel launches.
    //    
    // Note: As in the scan code, the calling code ensures that
    // allocated arrays are a power of 2 in size, so you can use your
    // exclusive_scan function with them. However, your implementation
    // must ensure that the results of find_repeats are correct given
    // the actual array length.

    int rounded_length = nextPow2(length);
    int blocks = (length + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
    int threadsPerBlock = blocks == 1 ? length : THREADS_PER_BLOCK;

    int* flags = nullptr;
    int* prefix = nullptr;
    hipMalloc(&flags, rounded_length * sizeof(int));
    hipMalloc(&prefix, rounded_length * sizeof(int));

    // input:  [1, 2, 2, 1, 1, 1, 3, 5, 3, 3]
    // flags:  [0, 1, 0, 1, 1, 0, 0, 0, 1, 0]
    // prefix: [0, 0, 1, 1, 2, 3, 3, 3, 3, 4]
    set_flag_knl<<<blocks, threadsPerBlock>>>(device_input, flags, length);
    hipDeviceSynchronize();
    exclusive_scan(flags, rounded_length, prefix);
    hipDeviceSynchronize();

    // index:  [0, 1, 2, 3, 4, 5, 6, 7, 8, 9]
    // flags:  [0, 1, 0, 1, 1, 0, 0, 0, 1, 0]
    // prefix: [_, 0, _, 1, 2, _, _, _, 3, _]
    // output: [1, 3, 4, 8]
    collect_index_knl<<<blocks, threadsPerBlock>>>(flags, prefix, device_output, length);
    hipDeviceSynchronize();

    int repeat_count = 0;
    hipMemcpy(&repeat_count, prefix + length - 1, sizeof(int), hipMemcpyDeviceToHost);

    hipFree(flags);
    hipFree(prefix);
    return repeat_count; 
}


//
// cudaFindRepeats --
//
// Timing wrapper around find_repeats. You should not modify this function.
double cudaFindRepeats(int *input, int length, int *output, int *output_length) {

    int *device_input;
    int *device_output;
    int rounded_length = nextPow2(length);
    
    hipMalloc((void **)&device_input, rounded_length * sizeof(int));
    hipMalloc((void **)&device_output, rounded_length * sizeof(int));
    hipMemcpy(device_input, input, length * sizeof(int), hipMemcpyHostToDevice);

    hipDeviceSynchronize();
    double startTime = CycleTimer::currentSeconds();
    
    int result = find_repeats(device_input, length, device_output);

    hipDeviceSynchronize();
    double endTime = CycleTimer::currentSeconds();

    // set output count and results array
    *output_length = result;
    hipMemcpy(output, device_output, length * sizeof(int), hipMemcpyDeviceToHost);

    hipFree(device_input);
    hipFree(device_output);

    float duration = endTime - startTime; 
    return duration;
}



void printCudaInfo()
{
    int deviceCount = 0;
    hipError_t err = hipGetDeviceCount(&deviceCount);

    printf("---------------------------------------------------------\n");
    printf("Found %d CUDA devices\n", deviceCount);

    for (int i=0; i<deviceCount; i++)
    {
        hipDeviceProp_t deviceProps;
        hipGetDeviceProperties(&deviceProps, i);
        printf("Device %d: %s\n", i, deviceProps.name);
        printf("   SMs:        %d\n", deviceProps.multiProcessorCount);
        printf("   Global mem: %.0f MB\n",
               static_cast<float>(deviceProps.totalGlobalMem) / (1024 * 1024));
        printf("   CUDA Cap:   %d.%d\n", deviceProps.major, deviceProps.minor);
    }
    printf("---------------------------------------------------------\n"); 
}
